#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#define GLM_FORCE_CUDA
#include "mj_raytracer_cuda.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>


// https://gamedev.stackexchange.com/a/146362
static inline __device__ float IntersectRayAABB(const mj::rt::Ray& ray, const glm::vec3& min, const glm::vec3& max)
{
  glm::vec3 dirInv = 1.0f / ray.direction;

  float t1 = (min.x - ray.origin.x) * dirInv.x;
  float t2 = (max.x - ray.origin.x) * dirInv.x;

  float tmin = fminf(t1, t2);
  float tmax = fmaxf(t1, t2);

  t1 = (min.y - ray.origin.y) * dirInv.y;
  t2 = (max.y - ray.origin.y) * dirInv.y;

  tmin = fmaxf(tmin, fminf(t1, t2));
  tmax = fminf(tmax, fmaxf(t1, t2));

  t1 = (min.z - ray.origin.z) * dirInv.z;
  t2 = (max.z - ray.origin.z) * dirInv.z;

  tmin = fmaxf(tmin, fminf(t1, t2));
  tmax = fminf(tmax, fmaxf(t1, t2));

  if (tmax >= tmin)
  {
    return tmin;
  }
  else
  {
    return -1.0f;
  }
}

static inline __device__ float IntersectRaySphere(const mj::rt::Ray& ray, const mj::rt::Sphere& sphere)
{
  glm::vec3 m = ray.origin - sphere.origin;
  float b = glm::dot(m, ray.direction);
  float c = glm::dot(m, m) - sphere.radius * sphere.radius;
  if (c > 0.0f && b > 0.0f)
  {
    return -1.0f;
  }
  float determinant = b * b - c;
  if (determinant < 0)
  {
    return -1.0f;
  }
  else
  {
    return -b - glm::sqrt(determinant);
  }
}

static inline __device__ float IntersectRayPlane(const mj::rt::Ray& ray, const mj::rt::Plane& plane)
{
  return -(glm::dot(ray.origin, plane.normal) + plane.distance) / glm::dot(ray.direction, plane.normal);
}

static inline __device__ glm::vec2 PixelToNDCSpace(uint16_t x, uint16_t y, uint16_t width, uint16_t height)
{
  return glm::vec2(((float) x + 0.5f) / width,
    ((float) y + 0.5f) / height);
}

// aspect is x/y
static inline __device__ glm::vec2 NDCToScreenSpace(const glm::vec2& ndc, float aspect)
{
  return glm::vec2((2.0f * ndc.x - 1.0f) * aspect, 1.0f - 2.0f * ndc.y);
}

// fov is in radians
static inline __device__ glm::vec2 ScreenToCameraSpace(const glm::vec2& ss, float fov)
{
  return ss * glm::tan(fov * 0.5f);
}

/**
 * @brief      Primary ray tracing kernel.
 *
 * @param      surface    The surface
 * @param[in]  width      The width
 * @param[in]  height     The height
 * @param[in]  pitch      The pitch
 * @param[in]  pConstant  The constant
 */
__global__ void cuda_raytracer(unsigned char* surface, int width, int height, size_t pitch, const mj::cuda::Constant* __restrict__ pConstant)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  float* pixel;

  if (x >= width || y >= height) return;

  // get a pointer to the pixel at (x,y)
  pixel = (float*) (surface + y * pitch) + 4 * x;

  glm::vec2 ndc = PixelToNDCSpace(x, y, width, height);
  glm::vec2 ss = NDCToScreenSpace(ndc, (float) width / height);
  glm::vec2 cs = ScreenToCameraSpace(ss, glm::radians(45.0f));

  mj::rt::Ray ray;
  glm::vec3 p = pConstant->mat * glm::vec4(cs, 1, 1);
  ray.origin = pConstant->s_Camera.position;
  ray.length = FLT_MAX;
  ray.direction = glm::normalize(p - pConstant->s_Camera.position);

  const mj::rt::Shape* pShape = nullptr;
  float t = FLT_MAX;
  for (const auto& shape : pConstant->s_Shapes)
  {
    switch (shape.type)
    {
    case mj::rt::Shape::Shape_Sphere:
      t = IntersectRaySphere(ray, shape.sphere);
      break;
    case mj::rt::Shape::Shape_Plane:
      t = IntersectRayPlane(ray, shape.plane);
      break;
    case mj::rt::Shape::Shape_AABB:
      t = IntersectRayAABB(ray, shape.aabb.min, shape.aabb.max);
      break;
    case mj::rt::Shape::Shape_Octree:
      // TODO
      break;
    default:
      break;
    }
    if (t >= 0.0f && t < ray.length)
    {
      ray.length = t;
      pShape = &shape;
    }
  }
  if (pShape)
  {
    glm::vec3 light = glm::normalize(glm::vec3(0.3f, 0.6f, -1.0f));

    // Get intersection normal
    glm::vec3 normal = glm::zero<glm::vec3>();
    const glm::vec3 intersection = ray.origin + ray.length * ray.direction;
    switch (pShape->type)
    {
    case mj::rt::Shape::Shape_Sphere:
      normal = (intersection - pShape->sphere.origin) / pShape->sphere.radius;
      break;
    case mj::rt::Shape::Shape_Plane:
      normal = pShape->plane.normal;
      break;
    case mj::rt::Shape::Shape_AABB:
    {
      // https://blog.johnnovak.net/2016/10/22/the-nim-raytracer-project-part-4-calculating-box-normals/
      glm::vec3 c = (pShape->aabb.min + pShape->aabb.max) * 0.5f; // aabb center
      glm::vec3 p = intersection - c; // vector from intersection to center
      glm::vec3 d = (pShape->aabb.min - pShape->aabb.max) * 0.5f; //??
      float bias = 1.0001f;

      normal = glm::normalize(glm::vec3((float) ((int) (p.x / glm::abs(d.x) * bias)),
        (float) ((int) (p.y / glm::abs(d.y) * bias)),
        (float) ((int) (p.z / glm::abs(d.z) * bias))));
    }
    break;
    case mj::rt::Shape::Shape_Octree:
      // TODO
      break;
    default:
      break;
    }

    glm::vec3 color = pShape->color;
    color *= glm::clamp(glm::dot(normal, light), 0.0f, 1.0f);
    color = glm::sqrt(color);
    pixel[0] = color.x;
    pixel[1] = color.y;
    pixel[2] = color.z;
    pixel[3] = 1.0f;
  }
  else
  {
    pixel[0] = 0.0f;
    pixel[1] = 0.0f;
    pixel[2] = 0.0f;
    pixel[3] = 1.0f;
  }
}

extern "C"
void cuda_texture_2d(void* surface, int width, int height, size_t pitch, const mj::cuda::Constant* pConstant)
{
  hipError_t error = hipSuccess;

  dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
  dim3 Dg = dim3((width + Db.x - 1) / Db.x, (height + Db.y - 1) / Db.y);

  cuda_raytracer << <Dg, Db >> > ((unsigned char*) surface, width, height, pitch, pConstant);

  error = hipGetLastError();

  if (error != hipSuccess)
  {
    printf("cuda_kernel_texture_2d() failed to launch error = %d\n", error);
  }
}
